#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "util.h"
#include "omp.h"
#include "cuda-nw.h"
#include "global.h"
using namespace std;

#define get_tid (threadIdx.x+blockIdx.x*blockDim.x)

/**
  * 打印矩阵
  * m 行, n 列
  */
__device__
void printMatrix(short *matrix, int m, int n) {
    for(int i=0;i<m;i++) {
        for(int j=0;j<n;j++)
            printf("%d ", matrix[i*m+j]);
        printf("\n");
    }
}

__device__
short max(short v1, short v2) {
    return v1 > v2 ? v1 : v2;
}
__device__
short max(short v1, short v2, short v3) {
    return max(max(v1, v2), v3);
}

__device__
int cuda_strlen(char *str) {
    int count = 0;
    while(str[count]!='\0')
        count++;
    return count;
}


/**
  * 每个线程有一个指向matrix的指针
  * matrix是一维的，大小是sizeof(short) * (m+1) * (maxLength+1)
  * 在堆中动态分配，每个kernel重复使用即可
  */
__device__ short* d_matrixPtr[MAX_THREADS * MAX_BLOCKS];

__global__
void allocDeviceMatrix(int centerSeqLength, int maxLength) {
    d_matrixPtr[get_tid] = new short[(centerSeqLength+1) * (maxLength+1)];
}

__global__
void freeDeviceMatrix() {
    if(d_matrixPtr[get_tid])
        delete[] d_matrixPtr[get_tid];
}



/**
  * m               in, 中心串长度
  * n               in, 对比串长度
  * centerSeq       in, 中心串
  * seqs            in, 其他n-1条串
  * seqIdx          in, 要被计算的串的编号
  * matrix          out, 需要计算的DP矩阵
  * 此函数没有被使用
  */
__device__
void cuda_nw(int m, int n, char *centerSeq, char *seq, short*matrix, int width) {
    // 初始化矩阵, DP矩阵m+1行,n+1列
    for(int i=0;i<=m;i++)
        matrix[i*width+0] = i * MISMATCH;   // matrix[i][0]
    for(int j=0;j<=n;j++)
        matrix[0*width+j] = j * MISMATCH;   // matrix[0][j];

    for(int i=1;i<=m;i++) {
        for(int j=1;j<=n;j++) {
            short up = matrix[(i-1)*width+j] + GAP;           // matrix[i-1][j]
            short left = matrix[i*width+j-1] + GAP;           // matrix[i][j-1]
            short diag = matrix[(i-1)*width+j-1] + ((centerSeq[i-1]==seq[j-1])?MATCH:MISMATCH);      // matrix[i-1][j-1]
            matrix[i*width+j] = max(up, left, diag);
        }
    }
}

#define COL_STEP 12
__device__
void cuda_nw_3d(int m, int n, char *centerSeq, char *seq, hipPitchedPtr matrix3DPtr) {
    size_t slicePitch = matrix3DPtr.pitch * (m+1);
    char *slice = (char *)matrix3DPtr.ptr + get_tid * slicePitch;

    // 初始化矩阵, DP矩阵m+1行,n+1列
    DPCell *matrixRow;
    for(int i=0;i<=m;i++) {
        matrixRow = (DPCell *)(slice + i * matrix3DPtr.pitch);
        matrixRow[0].score = MIN_SCORE;   // matrix[i][0]
        matrixRow[0].x_gap = MIN_SCORE;
        matrixRow[0].y_gap = GAP_START + i * GAP_EXTEND;
    }
    matrixRow = (DPCell *)(slice + 0 * matrix3DPtr.pitch);
    for(int j=0;j<=n;j++) {
        matrixRow[j].score = MIN_SCORE;   // matrix[0][j];
        matrixRow[j].x_gap = GAP_START + j * GAP_EXTEND;
        matrixRow[j].y_gap = MIN_SCORE;
    }
    matrixRow[0].score = 0;             // matrix[0][0]


    /**
      * 参照这篇论文：
      * [IPDPS-2009]An Efficient Implementation Of Smith Waterman Algorithm On Gpu Using Cuda, For Massively Parallel Scanning Of Sequence Databases
      * 横向计算，每次计算COL_STEP列，理论上讲COL_STEP越大越好，取决与register per block的限制
      * 这样左侧依赖数据，以及一列（COL_STEP个cell）内的上侧依赖数据就可以存储在register中
      * 有效减少global memory访问次数。
      * TODO: 1. 对角线的global memory访问也可以节省掉
      *       2. 如果中心串的长度不能被COL_STEP整除怎么处理
      */
    short upScore, upYGap, diagScore;
    for(int i=1;i<=m;i+=COL_STEP) {
        // 直接这样生命没有把所有元素初始化为MIN_SCORE
        //short leftScore[COL_STEP] = {MIN_SCORE}, leftXGap[COL_STEP] = {MIN_SCORE};
        short leftScore[COL_STEP], leftXGap[COL_STEP];
        for(int tmp=0;tmp<COL_STEP;tmp++) {
            leftScore[tmp] = MIN_SCORE;
            leftXGap[tmp] = MIN_SCORE;
        }

        for(int j=1;j<=n;j++) {
            for(int k=0;k<COL_STEP;k++) {
                if(i+k>m) break;
                DPCell *matrixRow = (DPCell *)(slice + (i+k) * matrix3DPtr.pitch);
                DPCell *matrixLastRow = (DPCell *)(slice + (i-1+k) * matrix3DPtr.pitch);

                DPCell cell;            // 当前计算的cell
                if(k==0) {
                    upScore = matrixLastRow[j].score;
                    upYGap = matrixLastRow[j].y_gap;
                    diagScore = matrixLastRow[j-1].score;
                }

                cell.x_gap = max(GAP_START+GAP_EXTEND+leftScore[k], GAP_EXTEND+leftXGap[k]);
                cell.y_gap = max(GAP_START+GAP_EXTEND+upScore, GAP_EXTEND+upYGap);
                cell.score = diagScore + ((centerSeq[i+k-1]==seq[j-1])?MATCH:MISMATCH);               // matrix[i-1][j-1]
                cell.score = max(cell.x_gap, cell.y_gap, cell.score);

                // 更新当前列下一行cell计算所需要的数据
                upScore = cell.score;
                upYGap = cell.y_gap;
                diagScore = leftScore[k];
                // 更新当前行下一列cell计算所需要的数据
                leftScore[k] = cell.score;
                leftXGap[k] = cell.x_gap;

                matrixRow[j] = cell;    // 写入当前cell到Global Memory
            }
        }
    }
}

/**
  * m               in, 中心串长度, m 行
  * n               in, 对比串长度, n 列
  * seqIdx          in, 要被计算的串的编号
  * matrix          in, 本次匹配得到的DP矩阵
  * space           out, 需要计算的本次匹配给中心串引入的空格
  * spaceForOther   out, 需要计算的本次匹配给当前串引入的空格
  * 此函数没有被使用
  */
__device__
void cuda_backtrack(int m, int n, short* matrix, short *spaceRow, short *spaceForOtherRow, int width) {
    // 从(m, n) 遍历到 (0, 0)
    // DP矩阵的纬度是m+1, n+1
    int i = m, j = n;
    while(i!=0 || j!=0) {
        int score = matrix[i*width+j];                              // matrix[i][j]
        //printf("%d,%d:  %d\n", i, j, score);
        if(i > 0 && matrix[(i-1)*width+j] + GAP == score) {         // matrix[i-1][j]
            spaceForOtherRow[j]++;                                  // spaceForOther[seqIdx][j]
            i--;
        } else if(j > 0 && matrix[i*width+j-1] + GAP == score) {    // matrix[i][j-1]
            spaceRow[i]++;                                          // space[seqIdx][i]
            j--;
        } else {
            i--;
            j--;
        }
    }
}
__device__
void cuda_backtrack_3d(int m, int n, char *centerSeq, char *seq, hipPitchedPtr matrix3DPtr, short *spaceRow, short *spaceForOtherRow) {
    size_t slicePitch = matrix3DPtr.pitch * (m+1);
    char *slice = (char *)matrix3DPtr.ptr + get_tid * slicePitch;

    int i = m, j = n;
    while(i!=0 || j!=0) {
        DPCell *matrixRow = (DPCell *)(slice + i * matrix3DPtr.pitch);
        DPCell *matrixLastRow = (DPCell *)(slice + (i-1) * matrix3DPtr.pitch);
        int score = (centerSeq[i-1] == seq[j-1]) ? MATCH : MISMATCH;
        if(i>0 && j>0 && score+matrixLastRow[j-1].score == matrixRow[j].score) {
            i--;
            j--;
        } else {
            int k = 1;
            while(true) {
                DPCell *matrixLastKRow = (DPCell *)(slice + (i-k) * matrix3DPtr.pitch);
                if(i>=k && matrixRow[j].score == matrixLastKRow[j].score+GAP_START+GAP_EXTEND*k) {
                    spaceForOtherRow[j] += k;
                    i = i - k;
                    break;
                } else if(j>=k && matrixRow[j].score == matrixRow[j-k].score+GAP_START+GAP_EXTEND*k) {
                    spaceRow[i] += k;
                    j = j - k;
                    break;
                } else {
                    k++;
                }
            }
        }
    }
}


__global__
void kernel(int startSeqIdx, char *centerSeq, char *seqs, int centerSeqLength, int *seqsSize, hipPitchedPtr matrix3DPtr, short *space, short *spaceForOther, int maxLength, int workCount) {

    int tid = get_tid;
    int seqIdx = tid + startSeqIdx;
    if(seqIdx >= workCount) return;

    // 得到当前线程要计算的串
    int width = maxLength + 1;
    char *seq = seqs + width * seqIdx;

    //int m = cuda_strlen(centerSeq);
    //int n = cuda_strlen(seq);
    int m = centerSeqLength;
    int n = seqsSize[seqIdx];

    // 当前匹配的字符串所需要填的空格数组的位置
    short *spaceRow = space + tid * (m+1);
    short *spaceForOtherRow = spaceForOther + tid * width;

    // 计算使用的DP矩阵
    cuda_nw_3d(m, n, centerSeq, seq, matrix3DPtr);
    cuda_backtrack_3d(m, n,centerSeq, seq, matrix3DPtr, spaceRow, spaceForOtherRow);

    //printMatrix(spaceForOtherRow, 1, n+1);
}


/**
  * 支持多个GPU
  * workCount:int       需要由GPU执行的工作量，平均分给各个GPU
  * centerSeq:string    中心串
  * seqs:vector<string> 除中心串外的所有串
  * maxLength:int       所有串的最长长度
  */
void cuda_msa(int offset, int workCount, string centerSeq, vector<string> seqs, int maxLength, short *space, short *spaceForOther);
void multi_gpu_msa(int workCount, string centerSeq, vector<string> seqs, int maxLength, short *space, short *spaceForOther) {
    if(workCount<= 0) return;

    int GPU_NUM;
    hipGetDeviceCount(&GPU_NUM);
    //GPU_NUM = 1;
    int workload = workCount / GPU_NUM;

    for(int i = 0; i < GPU_NUM; i++) {
        hipSetDevice(i);
        if(i != GPU_NUM - 1) {
            cuda_msa(i*workload, workload, centerSeq, seqs, maxLength, space, spaceForOther);
        } else {                // 最后一块GPU还要做多做余数
            cuda_msa(i*workload, workload+(workCount%GPU_NUM), centerSeq, seqs, maxLength, space, spaceForOther);
        }
    }

    hipDeviceReset();
}


void cuda_msa(int offset, int workCount, string centerSeq, vector<string> seqs, int maxLength, short *space, short *spaceForOther) {

    int sWidth = centerSeq.size() + 1;      // d_space的宽度
    int soWidth = maxLength + 1;            // d_spaceForOther的宽度

    // 1. 将中心串传到GPU
    char *d_centerSeq;
    hipMalloc((void**)&d_centerSeq, sWidth * sizeof(char));
    hipMemcpy(d_centerSeq, centerSeq.c_str(), sWidth *sizeof(char), hipMemcpyHostToDevice);

    // 2. 将需要匹配串拼接成一个长串传到GPU
    char *d_seqs;
    hipMalloc((void**)&d_seqs, (maxLength+1)*workCount*sizeof(char));
    char *c_seqs = new char[(maxLength+1) * workCount];
    for(int i=0;i<workCount;i++) {
        char *p = &(c_seqs[i * (maxLength + 1)]);
        strcpy(p, seqs[i+offset].c_str());
    }
    hipMemcpy(d_seqs, c_seqs, (maxLength+1)*workCount*sizeof(char), hipMemcpyHostToDevice);
    delete[] c_seqs;


    // 3. 将要匹配的串的长度也计算好传给GPU，因为在GPU上计算长度比较慢
    int *seqsSize = new int[workCount];
    for(int i = 0; i < workCount; i++)
        seqsSize[i] = seqs[i+offset].size();
    int *d_seqsSize;
    hipMalloc((void**)&d_seqsSize, sizeof(int)*workCount);
    hipMemcpy(d_seqsSize, seqsSize, sizeof(int)*workCount, hipMemcpyHostToDevice);
    delete[] seqsSize;


    // 每个kernel计算SEQUENCES_PER_KERNEL条串
    int SEQUENCES_PER_KERNEL = BLOCKS * THREADS;
    int h = workCount < SEQUENCES_PER_KERNEL ? workCount : SEQUENCES_PER_KERNEL;

    // 给存储空格信息申请空间
    // d_space, d_spaceForOther 是循环利用的
    short *d_space, *d_spaceForOther;
    hipMalloc((void**)&d_space, h*sWidth*sizeof(short));
    hipMalloc((void**)&d_spaceForOther, h*soWidth*sizeof(short));


    // 分配一个3D的DP Matrix
    size_t freeMem, totalMem;
    hipPitchedPtr matrix3DPtr;
    hipExtent matrixSize = make_hipExtent(sizeof(DPCell) * soWidth, sWidth, SEQUENCES_PER_KERNEL);
    hipMalloc3D(&matrix3DPtr, matrixSize);
    hipMemGetInfo(&freeMem, &totalMem);
    printf("freeMem :%luMB, totalMem: %luMB\n", freeMem/1024/1024, totalMem/1024/1024);

    for(int i = 0; i <= workCount / SEQUENCES_PER_KERNEL; i++) {
        if(i==workCount/SEQUENCES_PER_KERNEL)
            h = workCount % SEQUENCES_PER_KERNEL;

        // 此次kernel计算的起始串的位置（是相对位置，相对自己计算的起始串的）
        int startIdx = i * SEQUENCES_PER_KERNEL;
        printf("%d. idx: %d, h: %d\n", i, startIdx+offset, h);

        hipMemset(d_space, 0, h*sWidth*sizeof(short));
        hipMemset(d_spaceForOther, 0, h*soWidth*sizeof(short));

        kernel<<<BLOCKS, THREADS>>>(startIdx, d_centerSeq, d_seqs, centerSeq.size(), d_seqsSize, matrix3DPtr, d_space, d_spaceForOther, maxLength, workCount);
        hipError_t err  = hipGetLastError();
        if ( hipSuccess != err )
            printf("Error: %d, %s\n", err, hipGetErrorString(err));

        // 将空格信息传回给CPU
        // TODO：使用Pipeline可以重叠数据传输和kernel计算
        int spaceIdx = startIdx + offset;
        hipMemcpy(space+spaceIdx*sWidth, d_space, h*sWidth*sizeof(short), hipMemcpyDeviceToHost);
        hipMemcpy(spaceForOther+spaceIdx*soWidth, d_spaceForOther, h*soWidth*sizeof(short), hipMemcpyDeviceToHost);
    }

    hipFree(d_space);
    hipFree(d_spaceForOther);
    hipFree(d_centerSeq);
    hipFree(d_seqs);
    hipFree(matrix3DPtr.ptr);
}

