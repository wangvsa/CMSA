#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <vector>
#include "util.h"
#include "omp.h"
using namespace std;

#define MISMATCH -1
#define MATCH 0
#define GAP -1


int maxLength;
string centerSeq;
vector<string> seqs;
char *c_seqs;


/**
  * 打印矩阵
  * m 行, n 列
  */
__device__
void printMatrix(int *matrix, int m, int n) {
    for(int i=0;i<m;i++) {
        for(int j=0;j<n;j++)
            printf("%d ", matrix[i*m+j]);
        printf("\n");
    }
}

__device__
int max(int v1, int v2, int v3) {
    return max(max(v1, v2), v3);
}

__device__
int cuda_strlen(char *str) {
    int count = 0;
    while(str[count]!='\0')
        count++;
    return count;
}

/**
  * centerSeq       in, 中心串
  * seqs            in, 其他n-1条串
  * seqIdx          in, 要被计算的串的编号
  * matrix          out, 需要计算的DP矩阵
  */
__device__
void cuda_nw(char *centerSeq, char *seq, int *matrix, int maxLength) {
    // m行, n列
    int m = cuda_strlen(centerSeq) + 1;
    int n = cuda_strlen(seq) + 1;
    int width = maxLength + 1;

    // 初始化矩阵
    for(int i=0;i<m;i++)
        matrix[i*width+0] = i * MISMATCH;   // matrix[i][0]
    for(int j=0;j<n;j++)
        matrix[0*width+j] = j * MISMATCH;   // matrix[0][j];

    for(int i=1;i<m;i++) {
        for(int j=1;j<n;j++) {
            int up = matrix[(i-1)*width+j] + GAP;           // matrix[i-1][j]
            int left = matrix[i*width+j-1] + GAP;           // matrix[i][j-1]
            int diag = matrix[(i-1)*width+j-1] + ((centerSeq[i-1]==seq[j-1])?MATCH:MISMATCH);      // matrix[i-1][j-1]

            matrix[i*width+j] = max(up, left, diag);
        }
    }
    //printMatrix(matrix, m, n);
}

/**
  * m               in, 中心串长度, m 行
  * n               in, 对比串长度, n 列
  * seqIdx          in, 要被计算的串的编号
  * matrix          in, 本次匹配得到的DP矩阵
  * space           out, 需要计算的本次匹配给中心串引入的空格
  * spaceForOther   out, 需要计算的本次匹配给当前串引入的空格
  */
__device__
void cuda_backtrack(int m, int n, int seqIdx, int *matrix, int *spaceRow, int *spaceForOtherRow, int maxLength) {

    int width = maxLength + 1;

    // 从(m, n) 遍历到 (0, 0)
    // DP矩阵的纬度是m+1, n+1
    int i = m, j = n;
    while(i!=0 || j!=0) {
        int score = matrix[i*width+j];                              // matrix[i][j]
        //printf("%d,%d:  %d\n", i, j, score);
        if(i > 0 && matrix[(i-1)*width+j] + GAP == score) {         // matrix[i-1][j]
            spaceForOtherRow[j]++;                                  // spaceForOther[seqIdx][j]
            i--;
        } else if(j > 0 && matrix[i*width+j-1] + GAP == score) {    // matrix[i][j-1]
            spaceRow[i]++;                                          // space[seqIdx][i]
            j--;
        } else {
            i--;
            j--;
        }
    }
}


__global__
void cuda_msa(char *centerSeq, char *seqs, int *matrix, int *space, int *spaceForOther, size_t pitch, int maxLength) {
    int seqIdx = threadIdx.x + blockIdx.x * blockDim.x;

    int *matrixRow = (int*)((char*)matrix + seqIdx * pitch);
    char *seq = seqs + (maxLength+1) * seqIdx;

    int m = cuda_strlen(centerSeq);
    int n = cuda_strlen(seq);

    // 当前匹配的字符串所需要填的空格数组
    int *spaceRow = space + (seqIdx * (m+1));
    int *spaceForOtherRow = spaceForOther + (seqIdx * (maxLength+1));

    printf("centerSeq: %s, seq: %s\n", centerSeq, seq);
    printf("seqIdx: %d, m: %d, n: %d\n", seqIdx, m, n);

    cuda_nw(centerSeq, seq, matrixRow, maxLength);

    cuda_backtrack(m, n, seqIdx, matrixRow, spaceRow, spaceForOtherRow, maxLength);
    //printf("\n");
    //printMatrix(spaceRow, 1, m+1);
    //printMatrix(spaceForOtherRow, 1, n+1);
}

/**
 * 输出MSA
 * 设共有n条串，平均长度m
 * 构造中心串复杂度为:O(nm)
 * 构造其他条串复杂度为:O(nm)
 */
void output(int *space, int *spaceForOther) {
    vector<string> allAlignedStrs;
    int width = centerSeq.size() + 1;

    // 将所有串添加的空格汇总到一个数组中
    // 然后给中心串插入空格
    string alignedCenter(centerSeq);
    vector<int> spaceForCenter(centerSeq.size()+1, 0);
    for(int pos = centerSeq.size(); pos >= 0; pos--) {
        int count = 0;
        for(int idx = 0; idx < seqs.size(); idx++)
            count = (space[idx*width+pos] > count) ? space[idx*width+pos] : count;
        spaceForCenter[pos] = count;
        if(spaceForCenter[pos] > 0)
            //printf("pos:%d, space:%d\n", pos, spaceForCenter[pos]);
            alignedCenter.insert(pos, spaceForCenter[pos], '-');
    }

    printf("\n\n%s\n", alignedCenter.c_str());
    allAlignedStrs.push_back(alignedCenter);

    width = maxLength + 1;
    for(int idx = 0; idx < seqs.size(); idx++) {
        int shift = 0;
        string alignedStr(seqs[idx]);
        // 先插入自己比对时的空格
        for(int pos = seqs[idx].size(); pos >= 0; pos--) {
            if(spaceForOther[idx*width+pos] > 0)
                alignedStr.insert(pos, spaceForOther[idx*width+pos], '-');
        }
        // 再插入其他串比对时引入的空格
        for(int pos = 0; pos < spaceForCenter.size(); pos++) {
            int num = spaceForCenter[pos] - space[idx*width+pos];
            alignedStr.insert(pos+shift, num, '-');
            shift += spaceForCenter[pos];
        }
        printf("%s\n", alignedStr.c_str());
        allAlignedStrs.push_back(alignedStr);
    }

    // 将结果写入文件
    writeFastaFile("/home/wangchen/source/CUDA/CUDA-MSA/src/output2.fasta", allAlignedStrs);
}



void init() {
    // 读入所有字符串
    // centerSeq, 图中的纵向，决定了行数m
    // seqs[idx], 图中的横向，决定了列数n
    seqs = readFastaFile("/home/wangchen/source/CUDA/CUDA-MSA/test.fasta");
    centerSeq = seqs[0];
    seqs.erase(seqs.begin());

    maxLength = centerSeq.size();
    for(int i=0;i<seqs.size();i++)
        if( maxLength < seqs[i].size())
            maxLength = seqs[i].size();
    printf("max length: %d\n", maxLength);

    c_seqs = new char[(maxLength+1) * seqs.size()];
    for(int i=0;i<seqs.size();i++) {
        char *p = &(c_seqs[i * (maxLength + 1)]);
        strcpy(p, seqs[i].c_str());
    }
}


int main() {

    init();

    // 将二维数组一维化, 每行对应一条串
    int *d_space;
    int *d_spaceForOther;
    size_t pitch;
    int width = maxLength + 1;
    int height = seqs.size();
    hipMalloc((void**)&d_space, height*(centerSeq.size()+1)*sizeof(int));
    hipMalloc((void**)&d_spaceForOther, height*width*sizeof(int));
    hipMemset(d_space, 0, height*(centerSeq.size()+1)*sizeof(int));
    hipMemset(d_spaceForOther, 0, height*width*sizeof(int));

    // 每条串的DP矩阵是一行
    int *d_matrix;
    hipMallocPitch((void**)&d_matrix, &pitch, width*(centerSeq.size()+1)*sizeof(int), height);

    // 给字符串分配空间
    char *d_centerSeq;
    hipMalloc((void**)&d_centerSeq, (centerSeq.size()+1) * sizeof(char));
    hipMemcpy(d_centerSeq, centerSeq.c_str(), (centerSeq.size()+1)*sizeof(char), hipMemcpyHostToDevice);
    char *d_seqs;
    hipMalloc((void**)&d_seqs, width*height*sizeof(char));
    hipMemcpy(d_seqs, c_seqs, width*height*sizeof(char), hipMemcpyHostToDevice);

    cuda_msa<<<1, seqs.size()>>>(d_centerSeq, d_seqs, d_matrix, d_space, d_spaceForOther, pitch, maxLength);


    int *space = new int[height*(centerSeq.size()+1)];
    int *spaceForOther = new int[height*width];
    hipMemcpy(space, d_space, (centerSeq.size()+1)*height*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(spaceForOther, d_spaceForOther, width*height*sizeof(int), hipMemcpyDeviceToHost);

    output(space, spaceForOther);

    hipFree(d_space);
    hipFree(d_spaceForOther);
    hipFree(d_matrix);
    hipFree(d_centerSeq);
    hipFree(d_seqs);

    delete c_seqs;
    delete space;
    delete spaceForOther;

    return 0;
}
